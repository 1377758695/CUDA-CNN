#include "hip/hip_runtime.h"
#include "cuTrasformation.cuh"
#include <math.h>
#include <stdio.h>
#include "cuMatrix.h"
#include "util.h"
#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include "Config.h"
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define GAUSSIAN_FIELD_SIZE (21) // strictly odd number
hiprandGenerator_t rand_generator_device;
const hiprandRngType_t generator_type = HIPRAND_RNG_PSEUDO_DEFAULT;

cuMatrix<double>* cuGaussianKernel;
cuMatrix<double>* cuDispH;
cuMatrix<double>* cuDispV;

float * cu_d_randonNumf;
double* cu_d_randomNum;
double* cu_h_randomNum;
double dElasticSigma   = 4.0;   // higher numbers are more smooth and less distorted; Simard uses 4.0
//double dElasticScaling = 3.4;  // higher numbers amplify the distortions; Simard uses 3.4 (sic, maybe 0.34)


int getRandomNumLen(int batch, int ImgSize)
{
	return batch * (ImgSize * ImgSize * 2 + 3);
}

/*
	�������ܣ�������˹�˲�
	�̷߳���<<<dim3(1),dim3(GAUSSIAN_FIELD_SIZE*GAUSSIAN_FIELD_SIZE)>>>
*/
 __global__ void g_createGaussianKernel(double* gaussian, double dElasticSigma, int ImgSize)
{
 	int iiMid = GAUSSIAN_FIELD_SIZE >> 1;
 	double doubleElasticSigma = dElasticSigma * dElasticSigma;
 	int row = threadIdx.x % ImgSize;
 	int col = threadIdx.x / ImgSize;
 	double val1 = 1.0 / (dElasticSigma * 2.0 * 3.1415926535897932384626433832795);
 	double val2 = (row-iiMid)*(row-iiMid) + (col-iiMid)*(col-iiMid);
 
 	gaussian[threadIdx.x] = val1 * exp(-1.0 * val2 / (2.0 * doubleElasticSigma));
}

void cuInitDistortionMemery(int batch, int ImgSize)
{
	hiprandStatus_t curandstatus;
	hipError_t cuStatus;
	cuGaussianKernel = new cuMatrix<double>(GAUSSIAN_FIELD_SIZE, GAUSSIAN_FIELD_SIZE, 1);
	if(GAUSSIAN_FIELD_SIZE * GAUSSIAN_FIELD_SIZE > MAX_THREADS)
	{
		printf("g_createGaussianKernel > MAX_THREADS\n");
		exit(0);
	}
	g_createGaussianKernel<<<dim3(1),dim3(GAUSSIAN_FIELD_SIZE * GAUSSIAN_FIELD_SIZE)>>>(
		cuGaussianKernel->devData,
		dElasticSigma,
		ImgSize);
	hipDeviceSynchronize();

	//cu_d_randomNum
	cuStatus = hipMalloc((void**)&cu_d_randomNum, sizeof(double) * getRandomNumLen(batch, ImgSize));
	if(cuStatus != hipSuccess)
	{
		printf("hipMalloc cu_d_randomNum fail\n");
		exit(0);
	}

	//cu_d_randonNumf
	cuStatus = hipMalloc((void**)&cu_d_randonNumf, sizeof(float) * getRandomNumLen(batch, ImgSize));
	if(cuStatus != hipSuccess)
	{
		printf("hipMalloc cu_d_randomNumf fail\n");
		exit(0);
	}

	//cu_h_randomNum
	cu_h_randomNum = (double*)malloc(sizeof(double) * getRandomNumLen(batch, ImgSize));
	if(!cu_h_randomNum)
	{
		printf("malloc cu_h_randomNum fail\n");
	}

	//hiprandCreateGenerator

	curandstatus = hiprandCreateGenerator(&rand_generator_device, generator_type);
	if(curandstatus != HIPRAND_STATUS_SUCCESS)
	{
		printf("hiprandCreateGenerator fail\n");
		exit(0);
	}

	cuDispV = new cuMatrix<double>(batch, ImgSize * ImgSize, 1);
	cuDispH = new cuMatrix<double>(batch, ImgSize * ImgSize, 1);
}


__global__ void g_getRandomUniform(float* r1, double* r2, int len)
{
	for(int i = 0; i < len; i += blockDim.x)
	{
		int id = i + threadIdx.x;
		if(id < len)
		{
			r2[id] = r1[id] * 2 - 1;
		}
	}
}

/*�̷߳��䣺dim3(batch),dim3(ImgSize,ImgSize)*/
__global__ void g_generateDistortionMap(
	double* _dispH,
	double* _dispV,
	double* rand, 
	double* gaussianKernel,
	double dElasticScaling, 
	double dMaxScaling,
	double dMaxRotation,
	int ImgSize)
{
	double* uniformH = rand + blockIdx.x;
	double* uniformV = rand + blockIdx.x + ImgSize * ImgSize;
	double* dispH = _dispH + ImgSize * ImgSize * blockIdx.x;
	double* dispV = _dispV + ImgSize * ImgSize * blockIdx.x;

	for(int is = 0; is < ImgSize * ImgSize; is += blockDim.x)
	{
		int idx = is + threadIdx.x;
		if(idx < ImgSize * ImgSize)
		{
			int row = idx / ImgSize;
			int col = idx % ImgSize;
			int iiMid = GAUSSIAN_FIELD_SIZE / 2;

			double fConvolvedH = 0.0;
			double fConvolvedV = 0.0;
			double fSampleH, fSampleV;

			double elasticScale = dElasticScaling;

			for(int xxx = 0; xxx < GAUSSIAN_FIELD_SIZE; ++xxx)
			{
				for(int yyy = 0; yyy < GAUSSIAN_FIELD_SIZE; ++yyy)
				{
					int xxxDisp = col - iiMid + xxx;
					int yyyDisp = row - iiMid + yyy;

					if(xxxDisp < 0 || xxxDisp >= ImgSize || 
						xxxDisp < 0 || yyyDisp >= ImgSize)
					{
						fSampleH = 0.0;
						fSampleV = 0.0;
					}
					else 
					{
						fSampleH = uniformH[yyyDisp * ImgSize + xxxDisp];
						fSampleV = uniformV[yyyDisp * ImgSize + xxxDisp];
					}


					fConvolvedH += fSampleH * gaussianKernel[yyy * GAUSSIAN_FIELD_SIZE + xxx];
					fConvolvedV += fSampleV * gaussianKernel[yyy * GAUSSIAN_FIELD_SIZE + xxx];
				}
			}

			dispH[row * ImgSize + col] = elasticScale * fConvolvedH;
			dispV[row * ImgSize + col] = elasticScale * fConvolvedV;

			__syncthreads();

			double dSFHoriz = dMaxScaling / 100.0 * rand[blockIdx.x];
			double dSFVert  = dMaxScaling / 100.0 * rand[blockIdx.x + 1];

			int iMid = ImgSize / 2;

			dispH[row * ImgSize + col] += dSFHoriz * (col - iMid);
			dispV[row * ImgSize + col] += dSFVert  * (row - iMid);

			__syncthreads();

			double angle = dMaxRotation * rand[blockIdx.x];
			//double angle = dMaxRotation;
			angle = angle * 3.1415926535897932384626433832795 / 180.0;

			double cosAngle = cos(angle);
			double sinAngle = sin(angle);

			double xx = row - iMid;
			double yy = col - iMid;

			dispH[row * ImgSize + col] += yy - yy * cosAngle - xx * sinAngle;
			dispV[row * ImgSize + col] += xx - xx * cosAngle + yy * sinAngle;
		}
	}
}

/*�̷߳��䣺dim3(batch, channels),dim3(ImgSize, Imgsize)*/
__global__ void g_applyDistortionMap(
	double** _inputs,
	double** _outputs,
	double* _dispH, 
	double* _dispV, 
	int ImgSize)
{
	int c = blockIdx.y;

	int ImgSize2 = ImgSize * ImgSize;
	double* input = _inputs[blockIdx.x] + ImgSize2 * c;
	double* output= _outputs[blockIdx.x]+ ImgSize2 * c;
	double* dispV = _dispV + blockIdx.x * ImgSize2;
	double* dispH = _dispH + blockIdx.x * ImgSize2;

	for(int is = 0; is < ImgSize * ImgSize; is += blockDim.x)
	{
		int idx = is + threadIdx.x;
		if(idx < ImgSize * ImgSize)
		{
			int row = idx / ImgSize;
			int col = idx % ImgSize;

			double sourceRow, sourceCol;
			double fracRow, fracCol;
			double w1, w2, w3, w4;
			double sourceValue;
			int sRow, sCol, sRowp1, sColp1;
			bool bSkipOutOfBounds;

			sourceRow = (double)row - dispV[row * ImgSize + col];
			sourceCol = (double)col - dispH[row * ImgSize + col];

			fracRow = sourceRow - (int)sourceRow;
			fracCol = sourceCol - (int)sourceCol;

			w1 = ( 1.0 - fracRow ) * ( 1.0 - fracCol );
			w2 = ( 1.0 - fracRow ) * fracCol;
			w3 = fracRow * ( 1.0 - fracCol );
			w4 = fracRow * fracCol;

			bSkipOutOfBounds = false;

			if ( ((int)sourceRow + 1) >= ImgSize )	bSkipOutOfBounds = true;
			if ( (int)sourceRow < 0 )				bSkipOutOfBounds = true;

			if ( ((int)sourceCol + 1) >= ImgSize )	bSkipOutOfBounds = true;
			if ( (int)sourceCol < 0 )				bSkipOutOfBounds = true;

			if ( bSkipOutOfBounds == false )
			{
				// the supporting pixels for the "phantom" source pixel are all within the 
				// bounds of the character grid.
				// Manufacture its value by bi-linear interpolation of surrounding pixels

				sRow = (int)sourceRow;
				sCol = (int)sourceCol;

				sRowp1 = sRow + 1;
				sColp1 = sCol + 1;

				while (sRowp1 >= ImgSize ) sRowp1 -= ImgSize;
				while (sRowp1 < 0 ) sRowp1 += ImgSize;

				while (sColp1 >= ImgSize ) sColp1 -= ImgSize;
				while (sColp1 < 0 ) sColp1 += ImgSize;

				// perform bi-linear interpolation

				sourceValue =	
					w1 * input[sRow   * ImgSize + sCol] +
					w2 * input[sRow   * ImgSize + sColp1] +
					w3 * input[sRowp1 * ImgSize + sCol] +
					w4 * input[sRowp1 * ImgSize + sColp1];
			}
			else
			{
				// At least one supporting pixel for the "phantom" pixel is outside the
				// bounds of the character grid. Set its value to "background"
				// "background" color in the -1 -> +1 range of inputVector
				sourceValue = -1.0;  
			}
			output[row * ImgSize + col] = sourceValue;
		}
	}
	
}

void cuApplyRandom(int batch, unsigned long long s, int ImgSize)
{
	hiprandStatus_t hiprandStatus;
	hipError_t cudasSatus;
	unsigned long long seed = s;
	hiprandStatus = hiprandSetPseudoRandomGeneratorSeed(rand_generator_device, seed);

	if(hiprandStatus != HIPRAND_STATUS_SUCCESS)
	{
		printf("hiprandSetPseudoRandomGeneratorSeed fail\n");
		exit(0);
	}
	
	hiprandGenerateUniform(rand_generator_device, cu_d_randonNumf, getRandomNumLen(batch, ImgSize));

	g_getRandomUniform<<<dim3(1),dim3(256)>>>(cu_d_randonNumf, cu_d_randomNum, getRandomNumLen(batch, ImgSize));
	hipDeviceSynchronize();
	getLastCudaError("g_getRandomUniform");

	g_generateDistortionMap<<<dim3(batch),dim3(512)>>>(cuDispH->devData,
		cuDispV->devData, cu_d_randomNum, cuGaussianKernel->devData,
		Config::instance()->getDistortion(),
		Config::instance()->getScale(),
		Config::instance()->getRotation(), ImgSize);
	hipDeviceSynchronize();
	getLastCudaError("g_generateDistortionMap");

}

void cuApplyDistortion(double**inputs, double**outputs, int batch, int ImgSize)
{
	g_applyDistortionMap<<<dim3(batch, Config::instance()->getChannels()),
		dim3(512)>>>(inputs,
		outputs, 
		cuDispH->devData,
		cuDispV->devData,
		ImgSize);
	hipDeviceSynchronize();
}

/*�̰߳���<<<dim3(batch, channels),dim3(ImgSize,ImgSize)>>>*/
 __global__ void g_applyCropMap(double**_inputs, double**_outputs, double* random, double crop, int ImgSize)
 {
	int c = blockIdx.y;
 	double* input = _inputs[blockIdx.x] + c * ImgSize * ImgSize;
 	double* output= _outputs[blockIdx.x]+ c * ImgSize * ImgSize;

 	int sx =(int)(((random[blockIdx.x]     + 1.0) / 2.0 * crop) + 0.499999);
 	int sy =(int)(((random[blockIdx.x + 1] + 1.0) / 2.0 * crop) + 0.499999);
 	int ex = sx + ImgSize - 1 - crop;
 	int ey = sy + ImgSize - 1 - crop;
	for(int is = 0; is < ImgSize * ImgSize; is += blockDim.x)
	{
		int idx = is + threadIdx.x;
		if(idx < ImgSize * ImgSize)
		{
			int x  = idx / ImgSize;
			int y  = idx % ImgSize;
			if(x >= sx && x <= ex && y >=sy && y <= ey)
			{
				output[x * ImgSize + y] = input[x * ImgSize + y];
			}
			else
			{
				output[x * ImgSize + y] = -1.0;
			}
		}
	}
 }


void cuApplyCrop(double**inputs, double**outputs, int batch, int ImgSize)
{
	g_applyCropMap<<<dim3(batch, Config::instance()->getChannels()),
		dim3(512)>>>(inputs, outputs, cu_d_randomNum, Config::instance()->getCrop(), ImgSize);
	hipDeviceSynchronize();
	getLastCudaError("g_applyCropMap");
}