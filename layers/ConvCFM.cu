#include "hip/hip_runtime.h"
#include "ConvCFM.h"
#include "../common/cuBase.h"
#include "../common/Config.h"
/*
*	blocks : dim3(batch, cuKernelScan[0], Config::instance()->getChannels()),
*	threads: dim3(min(convOutputSize * convOutputSize, 512));
*/
__global__ void g_ConvCFM_feedforward_1(
	double** arrayS,
	double** arrayW,
	double** arrayB,
	double* conv,
	int inputSize,
	int kernelSize,
	int padding,
	int convSize,
	int convArea,
	int batch,
	int k1Amount);
/*
*	blocks : dim3(batch, cuKernelScan[0], Config::instance()->getChannels()),
*	threads: dim3(min(convOutputSize * convOutputSize, 512));
*/
__global__ void g_ConvCFM_feedforward_2(
	double* pool1,
	double** arrayW,
	double** arrayB,
	double* conv2,
	int pool1Size,
	int kernelSize,
	int padding,
	int conv2Size,
	int k1Amount,
	int k2Amount,
	int pool1Area,
	int conv2Area,
	int numOfCFM);

/*
* blocks : dim3(batch, numOfCFM * kernelAmount2, Config::instance()->getChannels())
* threads: dim3(threadidx)
*/
__global__ void g_ConvCFM_backpropagation(
	double* _convDelta,
	double**_w,
	double* _poolDelta,
	int     _convOutputSize,
	int     _poolOutputSize,
	int     _kernelAmount1,
	int     _kernelAmount2,
	int     _kernelSize,
	int     _padding,
	int     _convDeltaArea,
	int     _poolDeltaArea,
	int numOfCFM);

/*
* blocks  : dim3(batch, cuKernelScan[cl], Config::instance()->getChannels()),
* threads : dim3(threadidx)
*/
__global__ void g_ConvCFM_wgrad_2(double* pool,
	double* convDelta,
	double* WgradTmp,
	int poolOutputSize,
	int convOutputSize,
	int kernelAmount1,
	int kernelAmount2,
	int kernelSize,
	int padding,
	int poolArea,
	int convDeltaArea,
	int wgradTmpArea,
	int numOfCMF);


/*
* blocks  : dim3(kernelAmount2, kernelSize * kernelSize, Config::instance()->getChannels()),
* threads : dim3(256)
* shared  : sizeof(double) * 256
*/
__global__ void g_ConvCFM_wgradAdd_2(
	double* WgradTmp,
	double** Wgrad,
	double** w,
	int kernelAmount1,
	int kernelAmount2,
	int kernelSize,
	int batch,
	int wgradTmpArea,
	int wgradArea,
	int wArea,
	double lambda,
	int numOfCFM
	);

/*
* blocks  : dim3(kernelAmount2, Config::instance()->getChannels())
* threads : dim3(256)
* shared  : sizeof(double) * 256
*/
__global__ void g_ConvCFM_Bgrad_2(double* delta,
	double** bgrad,
	int deltaSize,
	int kernelAmount2,
	int batch,
	int deltaArea);

/*
* blocks  : dim3(batch, cuKernelScan[cl], Config::instance()->getChannels()),
* threads : dim3(threadidx)
*/
__global__ void g_ConvCFM_wgrad_1(double** sArray,
	double* convDelta,
	double* WgradTmp,
	int imgSize,
	int convOutputSize,
	int kernelAmount2,
	int kernelSize,
	int padding,
	int sArrayArea,
	int convDeltaArea,
	int wgrapTmpArea);

/*
* <<<dim3(k1, kernelSize*kernelSize, channels), dim3(256)>>>
*/
__global__ void g_ConvCFM_wgradAdd_1(double* WgradTmp, double** Wgrad,
	double** w,
	int kernelAmount2,
	int kernelSize,
	int batch,
	double lambda,
	int wgradTmpArea,
	int wgradArea,
	int wArea);

/*
*blocks  : dim3(kernelAmount2, Config::instance()->getChannels())
*threads : dim3(256)
*shared  : sizeof(double) * 256
*/
__global__ void g_ConvCFM_Bgrad_1(double* delta,
	double** bgrad,
	int deltaSize,
	int kernelAmount2,
	int batch,
	int deltaArea,
	int bgradArea);


void ConvCFM::getCost(cuMatrix<double>*cost, int* y)
{
	g_getCost_3<<<dim3(amount), dim3(32), sizeof(double) * 32>>>(cost->devData, 
		w.m_devPoint, 
		lambda,
		kernelSize, 
		kernelSize);
	hipDeviceSynchronize();
	getLastCudaError("ConvCFM:getCost");
}

void ConvCFM::feedforward()
{
	if((inputs_1 == NULL && inputs_2 == NULL) || (inputs_1 != NULL && inputs_2 != NULL))
	{
		printf("ConvCFM init error\n");
		exit(0);
	}
	if(inputs_1){
		dim3 block = dim3(batch, amount, Config::instance()->getChannels());
		dim3 thread= dim3(min(outputDim * outputDim, 512));
		g_ConvCFM_feedforward_1<<<block, thread>>>(inputs_1->m_devPoint,
			w.m_devPoint, 
			b.m_devPoint,
			outputs->devData,
			inputDim,
			kernelSize,
			padding,
			outputDim,
			outputs->getArea(),
			batch,
			amount);
		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("convCFM:g_ConvCFM_feedforward_1");
	}
	else if(inputs_2){
		dim3 block = dim3(batch, amount, Config::instance()->getChannels());
		dim3 thread= dim3(min(outputDim * outputDim, 512));
		g_ConvCFM_feedforward_2<<<block, thread>>>(inputs_2->devData,
			w.m_devPoint,
			b.m_devPoint,
			outputs->devData,
			inputDim,
			kernelSize,
			padding,
			outputDim,
			inputAmount,
			outputAmount,
			inputs_2->getArea(),
			outputs->getArea(),
			cfm);
		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("convCFM::g_ConvCFM_feedforward_2");
	}
	else{
		printf("ConvCFM init error\n");
		exit(0);
	}

	if(NON_LINEARITY >= 0){
		dim3 thread = dim3(min(256, outputs->getLen()));
		dim3 block  = dim3(min(256, (outputs->getLen() + thread.x - 1) / thread.x));
		g_nonLinearity<<<block, thread>>>(
			outputs->devData, 
			outputs->getLen(),
			NON_LINEARITY);
		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("convCFM::g_nonLinearity");
	}
}

void ConvCFM::backpropagation()
{
	if((inputs_1 == NULL && inputs_2 == NULL) || (inputs_1 != NULL && inputs_2 != NULL))
	{
		printf("ConvCFM init error\n");
		exit(0);
	}

	if(NON_LINEARITY >= 0){
		dim3 thread = dim3(min(256, outputs->getLen()));
		dim3 block  = dim3(min(256, (outputs->getLen() + thread.x - 1) / thread.x));

		g_dnonLinearity<<<block, thread>>>(curDelta->devData,
			outputs->devData, curDelta->getLen(), Config::instance()->getNonLinearity());

		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("ConvCFM::g_dnonLinearity");
	}
	
	if(inputs_2){
		dim3 block = dim3(batch, cfm * outputAmount, Config::instance()->getChannels());
		dim3 thread= min(outputDim * outputDim, 512);

		preDelta->gpuClear();

		g_ConvCFM_backpropagation<<<block, thread>>>(
			curDelta->devData,
			w.m_devPoint,
			preDelta->devData,
			outputDim,
			inputDim,
			inputAmount,
			outputAmount,
			kernelSize,
			padding,
			curDelta->getArea(),
			preDelta->getArea(),
			cfm);
		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("ConvCFM::g_ConvCFM_backpropagation");
	}
}


void ConvCFM::getGrad()
{
	if((inputs_1 == NULL && inputs_2 == NULL) || (inputs_1 != NULL && inputs_2 != NULL))
	{
		printf("ConvCFM init error\n");
		exit(0);
	}
	if(inputs_1){
		dim3 block = dim3(batch, outputAmount, Config::instance()->getChannels());
		dim3 thread= min(kernelSize * kernelSize, 512);
		g_ConvCFM_wgrad_1<<<block, thread>>>(
			inputs_1->m_devPoint,
			curDelta->devData,
			wgradTmp->devData,
			inputDim,
			outputDim,
			amount,
			kernelSize,
			padding,
			inputDim * inputDim,
			curDelta->getArea(),
			wgradTmp->getArea());

		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("g_ConvCFM_wgrad_1");

		block = dim3(outputAmount, kernelSize * kernelSize, Config::instance()->getChannels());
		thread= dim3(256);
		g_ConvCFM_wgradAdd_1<<<block, thread,
			sizeof(double) * 256>>>(
			wgradTmp->devData,
			wgrad.m_devPoint,
			w.m_devPoint,
			outputAmount,
			kernelSize,
			batch,
			lambda,
			wgradTmp->getArea(),
			wgrad[0]->getArea(),
			w[0]->getArea());

		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("ConvCFM::getGrad::g_wgradAdd_1");

		block = dim3(amount, Config::instance()->getChannels());
		thread= dim3(256);
		g_ConvCFM_Bgrad_1<<<block,thread,sizeof(double) * 256>>>
			(curDelta->devData,
			bgrad.m_devPoint,
			outputDim,
			outputAmount,
			batch,
			curDelta->getArea(),
			bgrad[0]->getArea());

		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("ConvCFM::getGrad::g_ConvCFM_Bgrad_1");
	}
	else if(inputs_2){
		dim3 block = dim3(batch, cfm * outputAmount, Config::instance()->getChannels());
		dim3 thread= min(kernelSize * kernelSize, 512);

		g_ConvCFM_wgrad_2<<<block, thread>>>(inputs_2->devData,
			curDelta->devData,
			wgradTmp->devData,
			inputDim,
			outputDim,
			inputAmount,
			outputAmount,
			kernelSize,
			padding,
			inputs_2->getArea(),
			curDelta->getArea(),
			wgradTmp->getArea(),
			cfm
			);
		hipDeviceSynchronize();
		getLastCudaError("g_ConvCFM_wgrad_2");

		block = dim3(amount, kernelSize * kernelSize, Config::instance()->getChannels());
		thread= dim3(256);
		g_ConvCFM_wgradAdd_2<<<block, thread, sizeof(double) * 256>>>(wgradTmp->devData,
			wgrad.m_devPoint,
			w.m_devPoint,
			inputAmount,
			outputAmount,
			kernelSize,
			batch,
			wgradTmp->getArea(),
			wgrad[0]->getArea(),
			w[0]->getArea(),
			lambda,cfm);
		hipDeviceSynchronize();
		getLastCudaError("g_ConvCFM_wgradAdd_2");


		block = dim3(amount, Config::instance()->getChannels());
		thread= dim3(256);
		g_ConvCFM_Bgrad_2<<<block, thread, sizeof(double) * 256>>>(curDelta->devData,
			bgrad.m_devPoint,
			outputDim,
			amount,
			batch,
			curDelta->getArea());
		hipDeviceSynchronize();
		getLastCudaError("g_ConvCFM_Bgrad_2");
	}
	else 
	{
		printf("ConvCFM init error\n");
		exit(0);
	}
}

void ConvCFM::updateWeight()
{
	dim3 thread = min(256, w[0]->getLen());
	dim3 block  = amount;
	g_vecAdd<<<block, thread>>>(momentum_w.m_devPoint, wgrad.m_devPoint, w.m_devPoint,
		momentum_b.m_devPoint, bgrad.m_devPoint, b.m_devPoint,
		w[0]->getLen(), b[0]->getLen(), 
		Config::instance()->getMomentum(),
		Config::instance()->getLrate());
}

ConvCFM::ConvCFM(std::string name)
{
	m_name = name;
	ConfigConv* config = (ConfigConv*)Config::instance()->getLayerByName(m_name);
	if(config->m_input == std::string("data"))
	{
		inputs_1 = Layers::instance()->getInputs();
		inputs_2 = NULL;
		inputAmount = 1;
		amount = config->m_amount;
		outputAmount = amount;
		kernelSize = config->m_kernelSize;
		padding = config->m_padding;

		inputDim  = Config::instance()->getImageSize();
		outputDim = (inputDim - kernelSize + 1) + padding * 2;
		batch     = Config::instance()->getBatchSize();
		lambda = config->m_weightDecay;
		cfm = 1;
		NON_LINEARITY = Config::instance()->getNonLinearity();

		outputs  = new cuMatrix<double>(batch, outputAmount * outputDim * outputDim, Config::instance()->getChannels());
		curDelta = new cuMatrix<double>(batch, outputAmount * outputDim * outputDim, Config::instance()->getChannels());
		wgradTmp = new cuMatrix<double>(batch, cfm * outputAmount * kernelSize * kernelSize, Config::instance()->getChannels());
		preDelta = NULL;

		for(int i = 0; i < amount; i++){
			w.push_back(new cuMatrix<double>(kernelSize, kernelSize, Config::instance()->getChannels()));
			b.push_back(new cuMatrix<double>(1, 1, Config::instance()->getChannels()));
			wgrad.push_back(new cuMatrix<double>(kernelSize, kernelSize, Config::instance()->getChannels()));
			bgrad.push_back(new cuMatrix<double>(1, 1, Config::instance()->getChannels()));
		}
		w.toGpu();
		b.toGpu();
		wgrad.toGpu();
		bgrad.toGpu();

		for(int i = 0; i < amount; i++){
			momentum_w.push_back(new cuMatrix<double>(kernelSize, kernelSize, Config::instance()->getChannels()));
			momentum_b.push_back(new cuMatrix<double>(1, 1, Config::instance()->getChannels()));
		}
		momentum_w.toGpu();
		momentum_b.toGpu();
	}
	else {
		ConfigConv* config = (ConfigConv*)Config::instance()->getLayerByName(m_name);
		ConvLayerBase * preLayer = (ConvLayerBase*)Layers::instance()->get(config->m_input);

		inputs_1 = NULL;
		inputs_2 = preLayer->getOutputs();
		inputAmount = preLayer->outputAmount;
		amount = config->m_amount;
		outputAmount = amount;
		kernelSize = config->m_kernelSize;
		padding = config->m_kernelSize;

		inputDim  = preLayer->outputDim;
		outputDim = (inputDim + 1 - kernelSize) + padding * 2;
		batch     = Config::instance()->getBatchSize();
		lambda    = config->m_weightDecay;
		cfm = config->m_cfm;
		NON_LINEARITY = Config::instance()->getNonLinearity();
		
		outputs = new cuMatrix<double>(batch, outputAmount * outputDim * outputDim, Config::instance()->getChannels());
		curDelta = new cuMatrix<double>(batch, outputAmount * outputDim  * outputDim,  Config::instance()->getChannels());
		wgradTmp = new cuMatrix<double>(batch, cfm * outputAmount  * kernelSize * kernelSize, Config::instance()->getChannels());
		preDelta = preLayer->getCurDelta();

		for(int i = 0; i < amount; i++){
			w.push_back(new cuMatrix<double>(kernelSize, kernelSize, Config::instance()->getChannels()));
			b.push_back(new cuMatrix<double>(1, 1, Config::instance()->getChannels()));
			wgrad.push_back(new cuMatrix<double>(kernelSize, kernelSize, Config::instance()->getChannels()));
			bgrad.push_back(new cuMatrix<double>(1, 1, Config::instance()->getChannels()));
		}

		w.toGpu();
		b.toGpu();
		wgrad.toGpu();
		bgrad.toGpu();

		for(int i = 0; i < amount; i++){
			momentum_w.push_back(new cuMatrix<double>(kernelSize, kernelSize, Config::instance()->getChannels()));
			momentum_b.push_back(new cuMatrix<double>(1, 1, Config::instance()->getChannels()));
		}
		momentum_w.toGpu();
		momentum_b.toGpu();
	}
	this->initRandom();
	Layers::instance()->set(m_name, this);
}

void ConvCFM::save(FILE* file)
{
	for(int a = 0; a < amount; a++){
		
		w[a]->toCpu();
		b[a]->toCpu();

		for(int c = 0; c < w[a]->channels; c++){
			for(int i = 0; i < w[a]->rows; i++){
				for(int j = 0; j < w[a]->cols; j++){
					fprintf(file, "%lf ", w[a]->get(i, j, c));
				}
			}
		}

		for(int c = 0; c < b[a]->channels; c++){
			for(int i = 0; i < b[a]->rows; i++){
				for(int j = 0; j < b[a]->cols; j++){
					fprintf(file, "%lf ", b[a]->get(i, j, c));
				}
			}
		}
	}
}

void ConvCFM::clearMomentum()
{
	for(int i = 0; i < momentum_b.size(); i++){
		momentum_b[i]->gpuClear();
	}
	for(int i = 0; i < momentum_w.size(); i++){
		momentum_w[i]->gpuClear();
	}
}

void ConvCFM::initRandom()
{
	srand(clock());
	for(int i = 0; i < w.size(); i++){
		double epsilon = 0.1;
		for(int c = 0; c < Config::instance()->getChannels(); c++)
		{
			double r1 = 0.5 + 4.0 * (rand()) / RAND_MAX;
			double r2 = 0.5 + 4.0 * (rand()) / RAND_MAX;
			createGaussian(w[i]->hostData + c * w[i]->getArea(), r1,r2,
				kernelSize, kernelSize, 
				Config::instance()->getChannels(), 
				epsilon * 0.5 + epsilon * rand() / RAND_MAX);
		}
		w[i]->toGpu();
	}
}

void ConvCFM::initFromCheckpoint(FILE* file)
{
	double val = 0;
	for(int a = 0; a < amount; a++){
		for(int c = 0; c < w[a]->channels; c++){
			for(int i = 0; i < w[a]->rows; i++){
				for(int j = 0; j < w[a]->cols; j++){
					fscanf(file, "%lf", &val);
					w[a]->set(i, j, c, val);
				}
			}
		}

		for(int c = 0; c < b[a]->channels; c++){
			fscanf(file, "%lf", &val);
			b[a]->set(0, 0, c, val);
		}
		w[a]->toGpu();
		b[a]->toGpu();
	}
}

__global__ void g_ConvCFM_feedforward_1(
	double** arrayS,
	double** arrayW,
	double** arrayB,
	double* conv,
	int inputSize,
	int kernelSize,
	int padding,
	int convSize,
	int convArea,
	int batch,
	int k1Amount)
{
	int sp = blockIdx.x;
	int k  = blockIdx.y;
	int c  = blockIdx.z;

	int convSize2  = convSize * convSize;
	int inputSize2 = inputSize* inputSize;
	int kernelSize2= kernelSize * kernelSize;

	int convSkip  = convArea * c + (sp * k1Amount + k) * convSize2;

	double* curInput = arrayS[sp] + c * inputSize2;
	double* w        = arrayW[k]  + c * kernelSize2;
	double  b        = arrayB[k][c];

	double* curConv  = conv   + convSkip;

	/*convolution*/
	for(int tidx = 0; tidx < convSize2; tidx += blockDim.x)
	{
		int idx = tidx + threadIdx.x;
		if(idx < convSize2)
		{
			int x = idx / convSize;
			int y = idx % convSize;
			double val = 0.0;
			for(int i = 0; i < kernelSize; i++)
			{
				for(int j = 0; j < kernelSize; j++)
				{
					int xx = x + i - padding;
					int yy = y + j - padding;
					if(xx >= 0 && xx < inputSize && yy >= 0 && yy < inputSize)
						val += curInput[xx * inputSize + yy] * w[i * kernelSize + j];
				}
			}
			curConv[idx] = val + b;
		}
	}
}


/*
* function: get convolution layer and pooling output
* blocks  : dim3(batch, kernelAmount2, Config::instance()->getChannels()),
* threads : dim3(threadidx)
*/


__global__ void g_ConvCFM_feedforward_2(
	double* pool1,
	double** arrayW,
	double** arrayB,
	double* conv2,
	int pool1Size,
	int kernelSize,
	int padding,
	int conv2Size,
	int k1Amount,
	int k2Amount,
	int pool1Area,
	int conv2Area,
	int numOfCFM)
{
	int sp = blockIdx.x;
	int k2 = blockIdx.y;
	int c  = blockIdx.z;

	double* w  = arrayW[k2] + kernelSize * kernelSize * c;
	double  b  = arrayB[k2][c];

	int pool1Size2 = pool1Size * pool1Size;
	int conv2Size2 = conv2Size * conv2Size;

	int skip2 = sp * k2Amount + k2;

	double* cv2 = conv2
		+ conv2Area * c
		+ skip2 * conv2Size2;

	for(int tidx = 0; tidx < conv2Size2; tidx += blockDim.x)
	{
		int idx = tidx + threadIdx.x;
		if(idx < conv2Size2)
		{
			int x = idx / conv2Size;
			int y = idx % conv2Size;
			double val = 0.0;
			for(int k1 = 0; k1 < numOfCFM; k1++)
			{
				int kk1 = (k1 + k2) % k1Amount;
				double* pl1 = pool1
					+ pool1Area * c
					+ (sp * k1Amount + kk1) * pool1Size2;

				for (int i = 0; i < kernelSize; i++) {
					for (int j = 0; j < kernelSize; j++) {
						int xx = x + i - padding;
						int yy = y + j - padding;
						if(xx>= 0 && xx < pool1Size && yy >= 0 && yy < pool1Size)
							val += pl1[xx * pool1Size + yy] * w[i * kernelSize + j];
					}
				}
			}
			cv2[idx] = val + b;
		}
	}
}


/*
* blocks  : dim3(batch, cuKernelScan[cl], Config::instance()->getChannels())
* threads : dim3(threadidx)
*/
__global__ void g_ConvCFM_backpropagation(
	double* _convDelta,
	double**_w,
	double* _poolDelta,
	int     _convOutputSize,
	int     _poolOutputSize,
	int     _kernelAmount1,
	int     _kernelAmount2,
	int     _kernelSize,
	int     _padding,
	int     _convDeltaArea,
	int     _poolDeltaArea,
	int numOfCFM)
{
	int curSize = _convOutputSize;
	int wSize = _kernelSize;
	int nxtSize = _poolOutputSize;
	int k2 = blockIdx.y % _kernelAmount2;
	int k1 = blockIdx.y / _kernelAmount2;

	cuAssert(k1 < numOfCFM);

	int kk1= (k1 + k2) % _kernelAmount1;

	int s = blockIdx.x;
	int c = blockIdx.z;
	int curSize2 = curSize * curSize;
	int nxtSize2 = nxtSize * nxtSize;
	double* curDelta = _convDelta + c * _convDeltaArea
		+ curSize2 * (s * _kernelAmount2 + k2);
	double* nxtDelta = _poolDelta + c * _poolDeltaArea
		+ nxtSize2 * (s * _kernelAmount1 + kk1);


	double* w = _w[k2] + c * _kernelSize * _kernelSize;

	for (int tidx = 0; tidx < nxtSize2; tidx += blockDim.x) {
		int idx = tidx + threadIdx.x;
		if (idx < nxtSize2) {
			int i = idx / nxtSize;
			int j = idx % nxtSize;
			double val = 0.0;
			for (int x = 0; x < wSize; x++) {
				for (int y = 0; y < wSize; y++) {
					int cx = i + x - (wSize >> 1);
					int cy = j + y - (wSize >> 1);
					int wx = wSize - x - 1;
					int wy = wSize - y - 1;
					cx -= ((wSize >> 1) - _padding);
					cy -= ((wSize >> 1) - _padding);
					if(cx >= 0 && cx < curSize && cy >= 0 && cy < curSize){
						val += curDelta[cx * curSize + cy] * w[wx * wSize + wy];
					}
				}
			}
			atomicAdd(nxtDelta + idx, val);
		}
	}
}


/*
* blocks  : dim3(batch, cuKernelScan[cl], Config::instance()->getChannels()),
* threads : dim3(threadidx)
*/
__global__ void g_ConvCFM_wgrad_2(double* pool,
	double* convDelta,
	double* WgradTmp,
	int poolOutputSize,
	int convOutputSize,
	int kernelAmount1,
	int kernelAmount2,
	int kernelSize,
	int padding,
	int poolArea,
	int convDeltaArea,
	int wgradTmpArea,
	int numOfCMF)
{
	int c  = blockIdx.z;
	int s  = blockIdx.x;
	int k2 = blockIdx.y % kernelAmount2;
	int k1 = blockIdx.y / kernelAmount2;

	cuAssert(k1 < numOfCMF);

	int kk1= (k1 + k2) % kernelAmount1;

	int curSize = poolOutputSize;
	int wSize   = convOutputSize;
	int nxtSize = kernelSize;
	int curSize2 = curSize * curSize;
	int wSize2   = wSize   * wSize;
	int nxtSize2 = nxtSize * nxtSize;
	double* cur   = pool
		+ c * poolArea
		+ curSize2 * (s * kernelAmount1 + kk1);
	double* w     = convDelta
		+ c * convDeltaArea
		+ wSize2 * (s * kernelAmount2 + k2);
	double* nxt   = WgradTmp
		+ c * wgradTmpArea
		+ nxtSize2 * (s * numOfCMF * kernelAmount2 + k1 * kernelAmount2 + k2);

	for(int tidx = 0; tidx < nxtSize2; tidx += blockDim.x)
	{
		int idx = tidx + threadIdx.x;
		if(idx < nxtSize2)
		{
			int i = idx / nxtSize;
			int j = idx % nxtSize;
			double val = 0.0;
			for(int x = 0; x < wSize; x++)
			{
				for(int y = 0; y < wSize; y++)
				{
					int cx = i + x - padding;
					int cy = j + y - padding;
					if(cx >= 0 && cy >= 0 && cx < curSize && cy < curSize)
						val += cur[cx * curSize + cy] * w[x * wSize + y];
				}
			}
			nxt[idx] = val;
		}
	}
}
/*
* blocks  : dim3(kernelAmount2, kernelSize * kernelSize, Config::instance()->getChannels()),
* threads : dim3(256)
* shared  : sizeof(double) * 256
*/
__global__ void g_ConvCFM_wgradAdd_2(
	double* WgradTmp,
	double** Wgrad,
	double** w,
	int kernelAmount1,
	int kernelAmount2,
	int kernelSize,
	int batch,
	int wgradTmpArea,
	int wgradArea,
	int wArea,
	double lambda,
	int numOfCFM)
{
	extern __shared__ double _sum[];
	int k2 = blockIdx.x;
	int kid= blockIdx.y;
	int c  = blockIdx.z;
	_sum[threadIdx.x] = 0;
	__syncthreads();
	int kernelSize2 = kernelSize * kernelSize;
	int  tlen = batch * numOfCFM;
	for(int i = 0; i <  tlen; i += blockDim.x)
	{
		int idx = i + threadIdx.x;
		if(idx < tlen)
		{
			int s  = idx / numOfCFM;
			int k1 = idx % numOfCFM;

			int id =
				c * wgradTmpArea
				+ kernelSize2 * (s * numOfCFM * kernelAmount2 + k1* kernelAmount2 + k2)
				+ kid;
			_sum[threadIdx.x] += WgradTmp[id];
		}
	}
	__syncthreads();
	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < (len >> 1))
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = (len + 1) >> 1;
	}
	__syncthreads();
	if(threadIdx.x == 0)
	{
		Wgrad[k2][kid + c * wgradArea] = _sum[0] / batch + w[k2][kid + c * wArea] * lambda;
	}
}
/*
* blocks  : dim3(kernelAmount2, Config::instance()->getChannels())
* threads : dim3(256)
* shared  : sizeof(double) * 256
*/
__global__ void g_ConvCFM_Bgrad_2(double* delta,
	double** bgrad,
	int deltaSize,
	int kernelAmount2,
	int batch,
	int deltaArea)
{
	extern __shared__ double _sum[];
	int k2 = blockIdx.x;
	int c  = blockIdx.y;
	_sum[threadIdx.x] = 0.0;
	__syncthreads();
	int deltaSize2 = deltaSize * deltaSize;
	int tlen = batch * deltaSize2;
	for(int i = 0; i < tlen; i += blockDim.x)
	{
		int idx = i + threadIdx.x;
		if(idx < tlen)
		{
			int s  = idx / deltaSize2;//s,kernel1
			int t2 = idx % deltaSize2;//x,y
			int id =
				c * deltaArea
				+ deltaSize2 * (s * kernelAmount2 + k2)
				+ t2;
			_sum[threadIdx.x] += delta[id];
		}
	}
	__syncthreads();
	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < (len >> 1))
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = (len + 1) >> 1;
	}
	__syncthreads();
	if(threadIdx.x == 0)
	{
		bgrad[k2][c] = _sum[0] / batch;
	}
}
/*
* blocks  : dim3(batch, cuKernelScan[cl], Config::instance()->getChannels()),
* threads : dim3(threadidx)
*/
__global__ void g_ConvCFM_wgrad_1(double** sArray,
	double* convDelta,
	double* WgradTmp,
	int imgSize,
	int convOutputSize,
	int kernelAmount2,
	int kernelSize,
	int padding,
	int sArrayArea,
	int convDeltaArea,
	int wgrapTmpArea)
{
	int curSize = imgSize;
	int wSize   = convOutputSize;
	int nxtSize = kernelSize;
	int s = blockIdx.x;
	int k2= blockIdx.y;
	int c = blockIdx.z;
	int wSize2   = wSize * wSize;
	int nxtSize2 = nxtSize * nxtSize;
	double* cur  = sArray[s] + c * sArrayArea;
	double* w     = convDelta
		+ c * convDeltaArea
		+ wSize2 * (s * kernelAmount2 + k2);
	double* nxt   = WgradTmp
		+ c * wgrapTmpArea
		+ nxtSize2 * (s * kernelAmount2 + k2);
	for(int tidx = 0; tidx < nxtSize2; tidx += blockDim.x)
	{
		int idx = tidx + threadIdx.x;
		if(idx < nxtSize2)
		{
			int i = idx / nxtSize;
			int j = idx % nxtSize;
			double val = 0.0;
			for(int x = 0; x < wSize; x++)
			{
				for(int y = 0; y < wSize; y++)
				{
					int cx = i + x - padding;
					int cy = j + y - padding;
					if(cx >= 0 &&  cy >= 0 && cx < curSize && cy < curSize)
						val += cur[cx * curSize + cy] * w[x * wSize + y];
				}
			}
			nxt[idx] = val;
		}
	}
}
/*
* <<<dim3(k1, kernelSize*kernelSize, channels), dim3(256)>>>
*/
__global__ void g_ConvCFM_wgradAdd_1(double* WgradTmp, double** Wgrad,
	double** w,
	int kernelAmount2,
	int kernelSize,
	int batch,
	double lambda,
	int wgradTmpArea,
	int wgradArea,
	int wArea)
{
	extern __shared__ double _sum[];
	int k2 = blockIdx.x;
	int kid= blockIdx.y;
	int c  = blockIdx.z;
	int tid= threadIdx.x;
	_sum[threadIdx.x] = 0;
	__syncthreads();
	int kernelSize2 = kernelSize * kernelSize;
	int tlen = batch;
	for(int i = 0; i < tlen; i += blockDim.x)
	{
		int s = i + threadIdx.x;
		if(s < tlen)
		{
			int id =
				c * wgradTmpArea
				+ kernelSize2 * (s * kernelAmount2 + k2)
				+ kid;
			_sum[threadIdx.x] += WgradTmp[id];
		}
	}
	__syncthreads();
	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(tid < (len >> 1))
		{
			_sum[tid] += _sum[tid + skip];
		}
		len = (len + 1) >> 1;
	}
	__syncthreads();
	if(tid == 0)
	{
		Wgrad[k2][kid + c * wgradArea] = _sum[0] / batch + w[k2][kid + c * wArea] * lambda;
	}
}
/*
*blocks  : dim3(kernelAmount2, Config::instance()->getChannels())
*threads : dim3(256)
*shared  : sizeof(double) * 256
*/
__global__ void g_ConvCFM_Bgrad_1(double* delta,
	double** bgrad,
	int deltaSize,
	int kernelAmount2,
	int batch,
	int deltaArea,
	int bgradArea)
{
	extern __shared__ double _sum[];
	int k2 = blockIdx.x;
	int c  = blockIdx.y;
	_sum[threadIdx.x] = 0.0;
	__syncthreads();
	int deltaSize2 = deltaSize * deltaSize;
	int tlen = deltaSize2 * batch;
	for(int i = 0; i < tlen; i += blockDim.x)
	{
		int idx = i + threadIdx.x;
		if(idx < tlen)
		{
			int s  = idx / (deltaSize2);//s
			int t2 = idx % (deltaSize2);//x,y
			int id =
				deltaArea * c
				+ deltaSize2 * (s * kernelAmount2 + k2)
				+ t2;
			_sum[threadIdx.x] += delta[id];
		}
	}
	__syncthreads();
	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < (len >> 1))
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = (len + 1) >> 1;
	}
	__syncthreads();
	if(threadIdx.x == 0)
	{
		bgrad[k2][c] = _sum[0] / batch;
	}
}