#include "hip/hip_runtime.h"
#include "Pooling.h"
#include <vector>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include "../common/Config.h"
#include "../common/cuBase.h"


/*
* function: unPooling
*/
__global__ void g_backpropagation(
	int* pointX,
	int* pointY,
	double* _pool,
	double* _conv,
	int poolSize,
	int convSize, 
	int poolDeltalen);

__global__ void g_feedforward(
	double* conv,
	double* pool,
	int* pointX,
	int* pointY,
	int convSize,
	int poolSize,
	int poolingSkip,
	int poolingSize,
	int convArea,
	int poolArea,
	int batch,
	int kAmount,
	int NONLIN);

void Pooling::feedforward()
{
	dim3 block = dim3(batch, amount, Config::instance()->getChannels());
	dim3 thread= dim3(512);
	
	g_feedforward<<<block, thread>>>(
		inputs->devData,
		outputs->devData,
		pointX->devData,
		pointY->devData,
		inputDim,
		outputDim,
		skip,
		size,
		inputs->getArea(),
		outputs->getArea(),
		batch,
		amount,
		NON_LINEARITY);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("pooling feedforward");
}

void Pooling::backpropagation()
{
	preDelta->gpuClear();

	int curDeltalen = curDelta->getLen();
	dim3 block = dim3(std::min(512, (curDeltalen + 511) / 512));
	dim3 thread= dim3(512);

	g_backpropagation<<<block, thread>>>(pointX->devData,
		pointY->devData,
		curDelta->devData,
		preDelta->devData,
		outputDim,
		inputDim,
		curDeltalen);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("pooling backpropagation");
}

Pooling::Pooling(cuMatrix<double>* _inputs, int _size, int _skip,int _inputDim, int _amount, int _batch) : inputs(_inputs),
	size(_size),
	skip(_skip),
	inputDim(_inputDim),
	amount(_amount),
	batch(_batch),
	preDelta(NULL)
{	
	outputDim = (inputDim + skip - 1) / skip;
	int channels   = inputs->channels;

	outputs  = new cuMatrix<double>(batch, amount * outputDim * outputDim, channels);
	pointX   = new cuMatrix<int>   (batch, amount * outputDim * outputDim, channels);
	pointY   = new cuMatrix<int>   (batch, amount * outputDim * outputDim, channels);

	curDelta = new cuMatrix<double>(batch, amount * outputDim * outputDim, channels);

}

/*
*blocks : dim3(batch, cuKernelScan[0], Config::instance()->getChannels()),
*threads: dim3(min(convOutputSize * convOutputSize, 512));
*/

__global__ void g_feedforward(
	double* conv,
	double* pool,
	int* pointX,
	int* pointY,
	int convSize,
	int poolSize,
	int poolingSkip,
	int poolingSize,
	int convArea,
	int poolArea,
	int batch,
	int kAmount,
	int NONLIN)
{
	int sp = blockIdx.x;
	int k  = blockIdx.y;
	int c  = blockIdx.z;

	int convSize2  = convSize * convSize;
	int poolSize2  = poolSize * poolSize;

	int convSkip  = convArea * c + (sp * kAmount + k) * convSize2;
	int poolSkip  = poolArea * c + (sp * kAmount + k) * poolSize2;

	double* curConv  = conv   + convSkip;
	double* curPool  = pool   + poolSkip;
	int* px          = pointX + poolSkip;
	int* py          = pointY + poolSkip;

	/*pooling*/
	for(int tidx = 0; tidx < poolSize2; tidx += blockDim.x)
	{
		int idx = tidx + threadIdx.x;
		if(idx < poolSize2)
		{
			int x = idx / poolSize;
			int y = idx % poolSize;

			int curX = x * poolingSkip;
			int curY = y * poolingSkip;

			cuAssert(curX < convSize && curY < convSize);

			double _max = curConv[curX * convSize + curY];
			int lenx = min(convSize, (x + 1) * poolingSize);
			int leny = min(convSize, (y + 1) * poolingSize);

			for(int i = curX; i < lenx; i++)
			{
				for(int j = curY; j < leny; j++)
				{
					double val = curConv[i * convSize + j];
					if(_max < val){
						_max  = val;
						curX = i;
						curY = j;
					}
				}
			}
			px     [idx] = curX;
			py     [idx] = curY;
			curPool[idx] = _max;
		}
	}
}

/*
* function: unPooling
*/
__global__ void g_backpropagation(int* pointX, int* pointY,
	double* _pool, double* _conv,
	int poolSize, int convSize, int poolDeltalen)
{
	int poolSize2 = poolSize * poolSize;
	int convSize2 = convSize * convSize;
	for(int i = 0; i < poolDeltalen; i += gridDim.x * blockDim.x)
	{
		int id = i + blockDim.x * blockIdx.x + threadIdx.x;
		if(id < poolDeltalen)
		{
			int convId = id / poolSize2;
			int idx    = id % poolSize2;
			int poolSkip = poolSize2 * convId;
			int*       x = pointX  + poolSkip;
			int*       y = pointY  + poolSkip;
			double* pool = _pool   + poolSkip;
			double* conv = _conv   + convSize2 * convId;
			int    curX = x   [idx];
			int    curY = y   [idx];
			double curP = pool[idx];
			cuAssert(curX < convSize && curY < convSize);
			atomicAdd(conv + curX * convSize + curY, curP);
		}
	}
}