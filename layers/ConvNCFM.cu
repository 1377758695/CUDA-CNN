#include "hip/hip_runtime.h"
#include "ConvNCFM.h"
#include "../common/cuBase.h"
#include "../common/Config.h"
/*
*	blocks : dim3(batch, cuKernelScan[0], Config::instance()->getChannels()),
*	threads: dim3(min(convOutputSize * convOutputSize, 512));
*/
__global__ void g_ConvNCFM_feedforward_1(
	double** arrayS,
	double** arrayW,
	double** arrayB,
	double* conv,
	int inputSize,
	int kernelSize,
	int padding,
	int convSize,
	int convArea,
	int batch,
	int k1Amount);
/*
*	blocks : dim3(batch, cuKernelScan[0], Config::instance()->getChannels()),
*	threads: dim3(min(convOutputSize * convOutputSize, 512));
*/
__global__ void g_ConvNCFM_feedforward_2(
	double* pool1,
	double** arrayW,
	double** arrayB,
	double* conv2,
	int pool1Size,
	int kernelSize,
	int padding,
	int conv2Size,
	int k1Scan,
	int k2Scan,
	int k1Amount,
	int k2Amount,
	int pool1Area,
	int conv2Area);

/*
* blocks  : dim3(batch, cuKernelScan[cl], Config::instance()->getChannels())
* threads : dim3(threadidx)
*/
__global__ void g_ConvNCFM_backpropagation(
	double* _convDelta,
	double**_w,
	double* _poolDelta,
	int     _convOutputSize,
	int     _poolOutputSize,
	int     _kernelScan1,
	int     _kernelScan2,
	int     _kernelAmount1,
	int     _kernelAmount2,
	int     _kernelSize,
	int     _padding,
	int     _convDeltaArea,
	int     _poolDeltaArea);

/*
* blocks  : dim3(batch, cuKernelScan[cl], Config::instance()->getChannels()),
* threads : dim3(threadidx)
*/
__global__ void g_ConvNCFM_wgrad_2(double* pool,
	double* convDelta,
	double* WgradTmp,
	int poolOutputSize,
	int convOutputSize,
	int kernelScan1,
	int kernelScan2,
	int kernelAmount1,
	int kernelAmount2,
	int kernelSize,
	int padding,
	int poolArea,
	int convDeltaArea,
	int wgradTmpArea);

/*
* blocks  : dim3(kernelAmount2, kernelSize * kernelSize, Config::instance()->getChannels()),
* threads : dim3(256)
* shared  : sizeof(double) * 256
*/
__global__ void g_ConvNCFM_wgradAdd_2(
	double* WgradTmp, 
	double** Wgrad,
	double** w,
	int kernelScan1,
	int kernelScan2,
	int kernelAmount1,
	int kernelAmount2,
	int kernelSize,
	int batch,
	int wgradTmpArea,
	int wgradArea,
	int wArea,
	double lambda);

/*
* blocks  : dim3(kernelAmount2, Config::instance()->getChannels())
* threads : dim3(256)
* shared  : sizeof(double) * 256
*/
__global__ void g_ConvNCFM_Bgrad_2(double* delta,
	double** bgrad,
	int deltaSize,
	int kernelScan1,
	int kernelScan2,
	int kernelAmount1,
	int kernelAmount2,
	int batch,
	int deltaArea);

/*
* blocks  : dim3(batch, cuKernelScan[cl], Config::instance()->getChannels()),
* threads : dim3(threadidx)
*/
__global__ void g_ConvNCFM_wgrad_1(double** sArray,
	double* convDelta,
	double* WgradTmp,
	int imgSize,
	int convOutputSize,
	int kernelScan2,
	int kernelAmount1,
	int kernelSize,
	int padding,
	int sArrayArea,
	int convDeltaArea,
	int wgrapTmpArea);

/*
* <<<dim3(k1, kernelSize*kernelSize, channels), dim3(256)>>>
*/
__global__ void g_ConvNCFM_wgradAdd_1(double* WgradTmp, double** Wgrad,
	double** w,
	int kernelScan2,
	int kernelAmount2,
	int kernelSize,
	int batch,
	double lambda,
	int wgradTmpArea,
	int wgradArea,
	int wArea);

/*
*blocks  : dim3(kernelAmount2, Config::instance()->getChannels())
*threads : dim3(256)
*shared  : sizeof(double) * 256
*/
__global__ void g_ConvNCFM_Bgrad_1(double* delta,
	double** bgrad,
	int deltaSize,
	int kernelScan2,
	int kernelAmount2,
	int batch,
	int deltaArea);


void ConvNCFM::getCost(cuMatrix<double>*cost, int* y)
{
	g_getCost_3<<<dim3(amount), dim3(32), sizeof(double) * 32>>>(cost->getDev(), 
		w.m_devPoint, 
		lambda,
		kernelSize, 
		kernelSize);
	hipDeviceSynchronize();
	getLastCudaError("ConvNCFM:getCost");
}

void ConvNCFM::feedforward()
{
	if((inputs_1 == NULL && inputs_2 == NULL) || (inputs_1 != NULL && inputs_2 != NULL))
	{
		printf("ConvNCFM init error\n");
		exit(0);
	}
	if(inputs_1){
		dim3 block = dim3(batch, amount, Config::instance()->getChannels());
		dim3 thread= dim3(min(outputDim * outputDim, 512));
		g_ConvNCFM_feedforward_1<<<block, thread>>>(inputs_1->m_devPoint,
			w.m_devPoint, 
			b.m_devPoint,
			outputs->getDev(),
			inputDim,
			kernelSize,
			padding,
			outputDim,
			outputs->getArea(),
			batch,
			amount);
		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("convNCFM:g_ConvNCFM_feedforward_1");
	}
	else if(inputs_2){
		dim3 block = dim3(batch, outputAmount, Config::instance()->getChannels());
		dim3 thread= dim3(min(outputDim * outputDim, 512));
		g_ConvNCFM_feedforward_2<<<block, thread>>>(inputs_2->getDev(),
			w.m_devPoint,
			b.m_devPoint,
			outputs->getDev(),
			inputDim,
			kernelSize,
			padding,
			outputDim,
			inputAmount,
			outputAmount,
			inputAmount,
			amount,
			inputs_2->getArea(),
			outputs->getArea());
		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("convNCFM::g_ConvNCFM_feedforward_2");
	}
	else{
		printf("ConvNCFM init error\n");
		exit(0);
	}

	if(NON_LINEARITY >= 0){
		dim3 thread = dim3(min(256, outputs->getLen()));
		dim3 block  = dim3(min(256, (outputs->getLen() + thread.x - 1) / thread.x));
		g_nonLinearity<<<block, thread>>>(
			outputs->getDev(), 
			outputs->getLen(),
			NON_LINEARITY);
		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("convNCFM::g_nonLinearity");
	}
}

void ConvNCFM::backpropagation()
{
	if((inputs_1 == NULL && inputs_2 == NULL) || (inputs_1 != NULL && inputs_2 != NULL))
	{
		printf("ConvNCFM init error\n");
		exit(0);
	}

	if(NON_LINEARITY >= 0){
		dim3 thread = dim3(min(256, outputs->getLen()));
		dim3 block  = dim3(min(256, (outputs->getLen() + thread.x - 1) / thread.x));

		g_dnonLinearity<<<block, thread>>>(curDelta->getDev(),
			outputs->getDev(), curDelta->getLen(), NON_LINEARITY);

		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("ConvNCFM::g_dnonLinearity");
	}
	
	if(inputs_2){
		dim3 block = dim3(batch, outputAmount, Config::instance()->getChannels());
		dim3 thread= min(outputDim * outputDim, 512);
		
		preDelta->gpuClear();

		g_ConvNCFM_backpropagation<<<block, thread>>>(
			curDelta->getDev(),
			w.m_devPoint,
			preDelta->getDev(),
			outputDim,
			inputDim,
			inputAmount,
			outputAmount,
			inputAmount,
			amount,
			kernelSize,
			padding,
			curDelta->getArea(),
			preDelta->getArea());
		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("ConvNCFM::g_ConvNCFM_backpropagation");
	}
}


void ConvNCFM::getGrad()
{
	if((inputs_1 == NULL && inputs_2 == NULL) || (inputs_1 != NULL && inputs_2 != NULL))
	{
		printf("ConvNCFM init error\n");
		exit(0);
	}
	if(inputs_1){
		dim3 block = dim3(batch, outputAmount, Config::instance()->getChannels());
		dim3 thread= min(kernelSize * kernelSize, 512);
		g_ConvNCFM_wgrad_1<<<block, thread>>>(
			inputs_1->m_devPoint,
			curDelta->getDev(),
			wgradTmp->getDev(),
			inputDim,
			outputDim,
			outputAmount,
			inputAmount,
			kernelSize,
			padding,
			inputDim * inputDim,
			curDelta->getArea(),
			wgradTmp->getArea());

		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("ConvNCFM::getGrad::g_wgrad_1");

		block = dim3(outputAmount, kernelSize * kernelSize, Config::instance()->getChannels());
		thread= dim3(256);
		g_ConvNCFM_wgradAdd_1<<<block, thread,
			sizeof(double) * 256>>>(
			wgradTmp->getDev(),
			wgrad.m_devPoint,
			w.m_devPoint,
			outputAmount,
			amount,
			kernelSize,
			batch,
			lambda,
			wgradTmp->getArea(),
			wgrad[0]->getArea(),
			w[0]->getArea());

		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("ConvNCFM::getGrad::g_wgradAdd_1");

		block = dim3(amount, Config::instance()->getChannels());
		thread= dim3(256);
		g_ConvNCFM_Bgrad_1<<<block,thread,sizeof(double) * 256>>>
			(curDelta->getDev(),
			bgrad.m_devPoint,
			outputDim,
			outputAmount,
			amount,
			batch,
			curDelta->getArea());

		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("ConvNCFM::getGrad::g_ConvNCFM_Bgrad_1");
	}
	else if(inputs_2){
		dim3 block = dim3(batch, outputAmount, Config::instance()->getChannels());
		dim3 thread= min(kernelSize * kernelSize, 512);

		g_ConvNCFM_wgrad_2<<<block, thread>>>(inputs_2->getDev(),
			curDelta->getDev(),
			wgradTmp->getDev(),
			inputDim,
			outputDim,
			inputAmount,
			outputAmount,
			inputAmount,
			amount,
			kernelSize,
			padding,
			inputs_2->getArea(),
			curDelta->getArea(),
			wgradTmp->getArea()
			);
		hipDeviceSynchronize();
		getLastCudaError("g_ConvNCFM_wgrad_2");

		block = dim3(amount, kernelSize * kernelSize, Config::instance()->getChannels());
		thread= dim3(256);
		g_ConvNCFM_wgradAdd_2<<<block, thread, sizeof(double) * 256>>>(wgradTmp->getDev(),
			wgrad.m_devPoint,
			w.m_devPoint,
			inputAmount,
			outputAmount,
			inputAmount,
			amount,
			kernelSize,
			batch,
			wgradTmp->getArea(),
			wgrad[0]->getArea(),
			w[0]->getArea(),
			lambda);
		hipDeviceSynchronize();
		getLastCudaError("g_ConvNCFM_wgradAdd_2");


		block = dim3(amount, Config::instance()->getChannels());
		thread= dim3(256);
		g_ConvNCFM_Bgrad_2<<<block, thread, sizeof(double) * 256>>>(curDelta->getDev(),
			bgrad.m_devPoint,
			outputDim,
			inputAmount,
			outputAmount,
			inputAmount,
			amount,
			batch,
			curDelta->getArea());
		hipDeviceSynchronize();
		getLastCudaError("g_ConvNCFM_wgradAdd_2");
	}
	else 
	{
		printf("ConvNCFM init error\n");
		exit(0);
	}
}

void ConvNCFM::updateWeight()
{
	dim3 thread = min(256, w[0]->getLen());
	dim3 block  = amount;

	g_vecAdd<<<block, thread>>>(momentum_w.m_devPoint, wgrad.m_devPoint, w.m_devPoint,
		momentum_b.m_devPoint, bgrad.m_devPoint, b.m_devPoint,
		w[0]->getLen(), b[0]->getLen(), 
		Config::instance()->getMomentum(),
		Config::instance()->getLrate());
}


ConvNCFM::ConvNCFM(std::string name)
{
	m_name = name;
	ConfigConv* config = (ConfigConv*)Config::instance()->getLayerByName(m_name);
	if(config->m_input  == std::string("data")){
		inputs_1 = Layers::instance()->getInputs();
		inputs_2 = NULL;
		inputAmount = 1;
		amount = config->m_amount;
		outputAmount = amount;
		kernelSize = config->m_kernelSize;
		padding = config->m_padding;

		inputDim  = Config::instance()->getImageSize();
		outputDim = (inputDim - kernelSize + 1) + padding * 2;
		batch     = Config::instance()->getBatchSize();
		lambda    = config->m_weightDecay;
		NON_LINEARITY = config->m_nonLinearity;

		outputs  = new cuMatrix<double>(batch, outputAmount * outputDim * outputDim, Config::instance()->getChannels());
		curDelta = new cuMatrix<double>(batch, outputAmount * outputDim * outputDim, Config::instance()->getChannels());
		wgradTmp = new cuMatrix<double>(batch, outputAmount * kernelSize * kernelSize, Config::instance()->getChannels());
		preDelta = NULL;

		for(int i = 0; i < amount; i++){
			w.push_back(new cuMatrix<double>(kernelSize, kernelSize, Config::instance()->getChannels()));
			b.push_back(new cuMatrix<double>(1, 1, Config::instance()->getChannels()));
			wgrad.push_back(new cuMatrix<double>(kernelSize, kernelSize, Config::instance()->getChannels()));
			bgrad.push_back(new cuMatrix<double>(1, 1, Config::instance()->getChannels()));
		}
		w.toGpu();
		b.toGpu();
		wgrad.toGpu();
		bgrad.toGpu();

		for(int i = 0; i < amount; i++){
			momentum_w.push_back(new cuMatrix<double>(kernelSize, kernelSize, Config::instance()->getChannels()));
			momentum_b.push_back(new cuMatrix<double>(1, 1, Config::instance()->getChannels()));
		}
		momentum_w.toGpu();
		momentum_b.toGpu();
	}else 
	{
		ConfigConv* config = (ConfigConv*)Config::instance()->getLayerByName(m_name);
		ConvLayerBase * preLayer = (ConvLayerBase*)Layers::instance()->get(config->m_input);

		inputs_1 = NULL;
		inputs_2 = preLayer->getOutputs();
		inputAmount = preLayer->outputAmount;
		amount = config->m_amount;
		outputAmount = inputAmount * amount;
		kernelSize = config->m_kernelSize;
		padding = config->m_padding;

		inputDim  = preLayer->outputDim;
		outputDim = (inputDim + 1 - kernelSize) + padding * 2;
		batch     = Config::instance()->getBatchSize();
		lambda    = config->m_weightDecay;
		NON_LINEARITY = config->m_nonLinearity;

		outputs = new cuMatrix<double>(batch, outputAmount * outputDim * outputDim, Config::instance()->getChannels());

		preDelta = preLayer->getCurDelta();
		curDelta = new cuMatrix<double>(batch, outputAmount * outputDim  * outputDim,  Config::instance()->getChannels());
		wgradTmp = new cuMatrix<double>(batch, outputAmount * kernelSize * kernelSize, Config::instance()->getChannels());

		for(int i = 0; i < amount; i++){
			w.push_back(new cuMatrix<double>(kernelSize, kernelSize, Config::instance()->getChannels()));
			b.push_back(new cuMatrix<double>(1, 1, Config::instance()->getChannels()));
			wgrad.push_back(new cuMatrix<double>(kernelSize, kernelSize, Config::instance()->getChannels()));
			bgrad.push_back(new cuMatrix<double>(1, 1, Config::instance()->getChannels()));
		}

		w.toGpu();
		b.toGpu();
		wgrad.toGpu();
		bgrad.toGpu();

		for(int i = 0; i < amount; i++){
			momentum_w.push_back(new cuMatrix<double>(kernelSize, kernelSize, Config::instance()->getChannels()));
			momentum_b.push_back(new cuMatrix<double>(1, 1, Config::instance()->getChannels()));
		}
		momentum_w.toGpu();
		momentum_b.toGpu();
	}

	this->initRandom();
	Layers::instance()->set(m_name, this);
}

void ConvNCFM::clearMomentum()
{
	for(int i = 0; i < momentum_b.size(); i++){
		momentum_b[i]->gpuClear();
	}
	for(int i = 0; i < momentum_w.size(); i++){
		momentum_w[i]->gpuClear();
	}
}

void ConvNCFM::save(FILE* file)
{
	for(int a = 0; a < amount; a++){
		w[a]->toCpu();
		b[a]->toCpu();
		for(int c = 0; c < w[a]->channels; c++){
			for(int i = 0; i < w[a]->rows; i++){
				for(int j = 0; j < w[a]->cols; j++){
					fprintf(file, "%lf ", w[a]->get(i, j, c));
				}
			}
		}

		for(int c = 0; c < b[a]->channels; c++){
			fprintf(file, "%lf ", b[a]->get(0, 0, c));
		}
	}
}

void ConvNCFM::initRandom()
{
	srand(clock());
	double initW = Config::instance()->getLayerByName(m_name)->m_initW;


	//  	for(int i = 0; i < w.size(); i++){
	//  		initMatrix(w[i], initW);
	//  	}
// 	for(int i = 0; i < w.size(); i++){
// 		for(int j = 0; j < w[i]->getLen(); j++){
// 			w[i]->hostData[j] =  initW * (2.0 * rand() / RAND_MAX - 1.0);
// 			printf("%lf ", w[i]->hostData[j]);
// 		}printf("\n");
// 		w[i]->toGpu();
// 	}
	srand(clock());
	for(int i = 0; i < w.size(); i++){
		double epsilon = 0.1;
		for(int c = 0; c < Config::instance()->getChannels(); c++)
		{
			double r1 = 0.5 + 4.0 * (rand()) / RAND_MAX;
			double r2 = 0.5 + 4.0 * (rand()) / RAND_MAX;
			createGaussian(w[i]->getHost() + c * w[i]->getArea(), r1,r2,
				kernelSize, kernelSize, 
				Config::instance()->getChannels(), 
				epsilon * 0.5 + epsilon * rand() / RAND_MAX);
		}
		w[i]->toGpu();
	}
}

void ConvNCFM::initFromCheckpoint(FILE* file)
{
	double val = 0;
	for(int a = 0; a < amount; a++){
		for(int c = 0; c < w[a]->channels; c++){
			for(int i = 0; i < w[a]->rows; i++){
				for(int j = 0; j < w[a]->cols; j++){
					fscanf(file, "%lf", &val);
					w[a]->set(i, j, c, val);
				}
			}
		}

		for(int c = 0; c < b[a]->channels; c++){
			fscanf(file, "%lf", &val);
			b[a]->set(0, 0, c, val);
		}
		w[a]->toGpu();
		b[a]->toGpu();
	}
}

__global__ void g_ConvNCFM_feedforward_1(
	double** arrayS,
	double** arrayW,
	double** arrayB,
	double* conv,
	int inputSize,
	int kernelSize,
	int padding,
	int convSize,
	int convArea,
	int batch,
	int k1Amount)
{
	int sp = blockIdx.x;
	int k  = blockIdx.y;
	int c  = blockIdx.z;

	int convSize2  = convSize * convSize;
	int inputSize2 = inputSize* inputSize;
	int kernelSize2= kernelSize * kernelSize;

	int convSkip  = convArea * c + (sp * k1Amount + k) * convSize2;

	double* curInput = arrayS[sp] + c * inputSize2;
	double* w        = arrayW[k]  + c * kernelSize2;
	double  b        = arrayB[k][c];

	double* curConv  = conv   + convSkip;

	/*convolution*/
	for(int tidx = 0; tidx < convSize2; tidx += blockDim.x)
	{
		int idx = tidx + threadIdx.x;
		if(idx < convSize2)
		{
			int x = idx / convSize;
			int y = idx % convSize;
			double val = 0.0;
			for(int i = 0; i < kernelSize; i++)
			{
				for(int j = 0; j < kernelSize; j++)
				{
					int xx = x + i - padding;
					int yy = y + j - padding;
					if(xx >= 0 && xx < inputSize && yy >= 0 && yy < inputSize)
						val += curInput[xx * inputSize + yy] * w[i * kernelSize + j];
				}
			}
			curConv[idx] = val + b;
		}
	}
}


/*
*	blocks : dim3(batch, cuKernelScan[0], Config::instance()->getChannels()),
*	threads: dim3(min(convOutputSize * convOutputSize, 512));
*/

__global__ void g_ConvNCFM_feedforward_2(
	double* pool1,
	double** arrayW,
	double** arrayB,
	double* conv2,
	int pool1Size,
	int kernelSize,
	int padding,
	int conv2Size,
	int k1Scan,
	int k2Scan,
	int k1Amount,
	int k2Amount,
	int pool1Area,
	int conv2Area)
{
	int sp = blockIdx.x;
	int c  = blockIdx.z;
	int k2 = blockIdx.y % k2Amount;
	int k1 = blockIdx.y / k2Amount;

	double* w   = arrayW[k2] + kernelSize * kernelSize * c;
	double  b   = arrayB[k2][c];

	int pool1Size2 = pool1Size * pool1Size;
	int conv2Size2 = conv2Size * conv2Size;

	int skip1 = sp * k1Scan + k1;
	int skip2 = sp * k2Scan + k1 * k2Amount + k2;

	double* pl1 = pool1
		+ pool1Area * c
		+ skip1 * pool1Size2;

	double* cv2 = conv2
		+ conv2Area * c
		+ skip2 * conv2Size2;

	for(int tidx = 0; tidx < conv2Size2; tidx += blockDim.x)
	{
		int idx = tidx + threadIdx.x;
		if(idx < conv2Size2)
		{
			int x = idx / conv2Size;
			int y = idx % conv2Size;
			double val = 0.0;
			for(int i = 0; i < kernelSize; i++)
			{
				for(int j = 0; j < kernelSize; j++)
				{
					int xx = x + i - padding;
					int yy = y + j - padding;
					if(xx>= 0 && xx < pool1Size && yy >= 0 && yy < pool1Size)
						val += pl1[xx * pool1Size + yy] * w[i * kernelSize + j];
				}
			}
			cv2[idx] = val + b;
		}
	}
}



/*
* blocks  : dim3(batch, cuKernelScan[cl], Config::instance()->getChannels())
* threads : dim3(threadidx)
*/
__global__ void g_ConvNCFM_backpropagation(
	double* _convDelta,
	double**_w,
	double* _poolDelta,
	int     _convOutputSize,
	int     _poolOutputSize,
	int     _kernelScan1,
	int     _kernelScan2,
	int     _kernelAmount1,
	int     _kernelAmount2,
	int     _kernelSize,
	int     _padding,
	int     _convDeltaArea,
	int     _poolDeltaArea)  
{
	int curSize          = _convOutputSize;
	int wSize            = _kernelSize;
	int nxtSize          = _poolOutputSize;
	int k1 = blockIdx.y / _kernelAmount2;
	int k2 = blockIdx.y % _kernelAmount2;
	int s  = blockIdx.x;
	int c  = blockIdx.z;
	int curSize2 = curSize * curSize;
	int nxtSize2 = nxtSize * nxtSize;
	int skip1 = s * _kernelScan1 + k1;
	int skip2 = s * _kernelScan2 + k1 * _kernelAmount2 + k2;
	double* curDelta = _convDelta 
		+ c * _convDeltaArea
		+ skip2 * curSize2;
	double* nxtDelta = _poolDelta 
		+ c * _poolDeltaArea
		+ skip1 * nxtSize2;
	double*        w = _w[k2] + c * _kernelSize * _kernelSize;
	for(int tidx = 0; tidx < nxtSize2; tidx += blockDim.x)
	{
		int idx = tidx + threadIdx.x;
		if(idx < nxtSize2)
		{
			int i = idx / nxtSize;
			int j = idx % nxtSize;
			double val = 0.0;
			for(int x = 0; x < wSize; x++)
			{
				for(int y = 0; y < wSize; y++)
				{
					int cx = i + x - (wSize >> 1);
					int cy = j + y - (wSize >> 1);
					int wx = wSize - x - 1;
					int wy = wSize - y - 1;
					cx -= ((wSize >> 1) - _padding);
					cy -= ((wSize >> 1) - _padding);
					if(cx >= 0 && cx < curSize && cy >= 0 && cy < curSize){
						val += curDelta[cx * curSize + cy] * w[wx * wSize + wy];
					}
				}
			}
			atomicAdd(nxtDelta + idx, val);
		}
	}
}


/*
* blocks  : dim3(batch, cuKernelScan[cl], Config::instance()->getChannels()),
* threads : dim3(threadidx)
*/
__global__ void g_ConvNCFM_wgrad_2(double* pool,
	double* convDelta,
	double* WgradTmp,
	int poolOutputSize,
	int convOutputSize,
	int kernelScan1,
	int kernelScan2,
	int kernelAmount1,
	int kernelAmount2,
	int kernelSize,
	int padding,
	int poolArea,
	int convDeltaArea,
	int wgradTmpArea)
{
	int c = blockIdx.z;
	int s = blockIdx.x;
	int k2= blockIdx.y % kernelAmount2;
	int k1= blockIdx.y / kernelAmount2;
	int curSize = poolOutputSize;
	int wSize   = convOutputSize;
	int nxtSize = kernelSize;
	int curSize2 = curSize * curSize;
	int wSize2   = wSize   * wSize;
	int nxtSize2 = nxtSize * nxtSize;
	double* cur   = pool
		+ c * poolArea
		+ curSize2 * (s * kernelScan1 + k1);
	double* w     = convDelta
		+ c * convDeltaArea
		+ wSize2 * (s * kernelScan2 + k1* kernelAmount2 + k2);
	double* nxt   = WgradTmp
		+ c * wgradTmpArea
		+ nxtSize2 * (s * kernelScan2 + k1* kernelAmount2 + k2);
	for(int tidx = 0; tidx < nxtSize2; tidx += blockDim.x)
	{
		int idx = tidx + threadIdx.x;
		if(idx < nxtSize2)
		{
			int i = idx / nxtSize;
			int j = idx % nxtSize;
			double val = 0.0;
			for(int x = 0; x < wSize; x++)
			{
				for(int y = 0; y < wSize; y++)
				{
					int cx = i + x - padding;
					int cy = j + y - padding;
					if(cx >= 0 &&  cy >= 0 && cx < curSize && cy < curSize)
						val += cur[cx * curSize + cy] * w[x * wSize + y];
				}
			}
			nxt[idx] = val;
		}
	}
}
/*
* blocks  : dim3(kernelAmount2, kernelSize * kernelSize, Config::instance()->getChannels()),
* threads : dim3(256)
* shared  : sizeof(double) * 256
*/
__global__ void g_ConvNCFM_wgradAdd_2(
	double* WgradTmp, 
	double** Wgrad,
	double** w,
	int kernelScan1,
	int kernelScan2,
	int kernelAmount1,
	int kernelAmount2,
	int kernelSize,
	int batch,
	int wgradTmpArea,
	int wgradArea,
	int wArea,
	double lambda)
{
	extern __shared__ double _sum[];
	int k2 = blockIdx.x;
	int kid= blockIdx.y;
	int c  = blockIdx.z;
	_sum[threadIdx.x] = 0;
	__syncthreads();
	int kernelSize2 = kernelSize * kernelSize;
	int  tlen = batch * kernelScan1;
	for(int i = 0; i <  tlen; i += blockDim.x)
	{
		int idx = i + threadIdx.x;
		if(idx < tlen)
		{
			int s = idx / kernelScan1;
			int k1= idx % kernelScan1;
			int id = c * wgradTmpArea
				+ kernelSize2 * (s * kernelScan2 + k1* kernelAmount2 + k2) + kid;
			_sum[threadIdx.x] += WgradTmp[id];
		}
	}
	__syncthreads();
	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < (len >> 1))
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = (len + 1) >> 1;
	}
	__syncthreads();
	if(threadIdx.x == 0)
	{
		Wgrad[k2][kid + c * wgradArea] = _sum[0] / batch + w[k2][kid + c * wArea] * lambda;
	}
}
/*
* blocks  : dim3(kernelAmount2, Config::instance()->getChannels())
* threads : dim3(256)
* shared  : sizeof(double) * 256
*/
__global__ void g_ConvNCFM_Bgrad_2(double* delta,
	double** bgrad,
	int deltaSize,
	int kernelScan1,
	int kernelScan2,
	int kernelAmount1,
	int kernelAmount2,
	int batch,
	int deltaArea)
{
	extern __shared__ double _sum[];
	int k2 = blockIdx.x;
	int c  = blockIdx.y;
	_sum[threadIdx.x] = 0.0;
	__syncthreads();
	int deltaSize2 = deltaSize * deltaSize;
	int tlen = batch * kernelScan1 * deltaSize2;
	for(int i = 0; i < tlen; i += blockDim.x)
	{
		int idx = i + threadIdx.x;
		if(idx < tlen)
		{
			int t1 = idx / deltaSize2;//s,kernel1
			int t2 = idx % deltaSize2;//x,y
			int s  = t1 / kernelScan1;
			int k1 = t1 % kernelScan1;
			int id = 
				c * deltaArea
				+ deltaSize2 * (s * kernelScan2 + k1* kernelAmount2 + k2)
				+ t2;

			_sum[threadIdx.x] += delta[id];
		}
	}
	__syncthreads();
	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < (len >> 1))
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = (len + 1) >> 1;
	}
	__syncthreads();
	if(threadIdx.x == 0)
	{
		bgrad[k2][c] = _sum[0] / batch;
	}
}
/*
* blocks  : dim3(batch, cuKernelScan[cl], Config::instance()->getChannels()),
* threads : dim3(threadidx)
*/
__global__ void g_ConvNCFM_wgrad_1(double** sArray,
	double* convDelta,
	double* WgradTmp,
	int imgSize,
	int convOutputSize,
	int kernelScan2,
	int kernelAmount1,
	int kernelSize,
	int padding,
	int sArrayArea,
	int convDeltaArea,
	int wgrapTmpArea)
{
	int curSize = imgSize;
	int wSize   = convOutputSize;
	int nxtSize = kernelSize;
	int s  = blockIdx.x;
	int k2 = blockIdx.y;
	int c  = blockIdx.z;
	int wSize2   = wSize * wSize;
	int nxtSize2 = nxtSize * nxtSize;
	double* cur  = sArray[s] + c * sArrayArea;
	double* w     = convDelta
		+ c * convDeltaArea
		+ wSize2 * (s * kernelScan2 + k2);
	double* nxt   = WgradTmp
		+ c * wgrapTmpArea
		+ nxtSize2 * (s * kernelScan2 + k2);
	for(int tidx = 0; tidx < nxtSize2; tidx += blockDim.x)
	{
		int idx = tidx + threadIdx.x;
		if(idx < nxtSize2)
		{
			int i = idx / nxtSize;
			int j = idx % nxtSize;
			double val = 0.0;
			for(int x = 0; x < wSize; x++)
			{
				for(int y = 0; y < wSize; y++)
				{
					int cx = i + x - padding;
					int cy = j + y - padding;
					if(cx >= 0 && cy >= 0 && cx < curSize && cy < curSize)
						val += cur[cx * curSize + cy] * w[x * wSize + y];
				}
			}
			nxt[idx] = val;
		}
	}
}
/*
* <<<dim3(k1, kernelSize*kernelSize, channels), dim3(256)>>>
*/
__global__ void g_ConvNCFM_wgradAdd_1(double* WgradTmp, double** Wgrad,
	double** w,
	int kernelScan2,
	int kernelAmount2,
	int kernelSize,
	int batch,
	double lambda,
	int wgradTmpArea,
	int wgradArea,
	int wArea)
{
	extern __shared__ double _sum[];
	int k2 = blockIdx.x;
	int kid= blockIdx.y;
	int c  = blockIdx.z;
	int tid= threadIdx.x;
	_sum[threadIdx.x] = 0;
	__syncthreads();
	int kernelSize2 = kernelSize * kernelSize;
	int tlen = batch;
	for(int i = 0; i < tlen; i += blockDim.x)
	{
		int s = i + threadIdx.x;
		if(s < tlen)
		{
			int id = 
				c * wgradTmpArea
				+ kernelSize2 * s * kernelScan2
				+ kernelSize2 * k2 + kid;
			_sum[threadIdx.x] += WgradTmp[id];
		}
	}
	__syncthreads();
	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(tid < (len >> 1))
		{
			_sum[tid] += _sum[tid + skip];
		}
		len = (len + 1) >> 1;
	}
	__syncthreads();
	if(tid == 0)
	{
		Wgrad[k2][kid + c * wgradArea] = _sum[0] / batch + w[k2][kid + c * wArea] * lambda;
	}
}
/*
*blocks  : dim3(kernelAmount2, Config::instance()->getChannels())
*threads : dim3(256)
*shared  : sizeof(double) * 256
*/
__global__ void g_ConvNCFM_Bgrad_1(double* delta,
	double** bgrad,
	int deltaSize,
	int kernelScan2,
	int kernelAmount2,
	int batch,
	int deltaArea)
{
	extern __shared__ double _sum[];
	int k2 = blockIdx.x;
	int c  = blockIdx.y;
	_sum[threadIdx.x] = 0.0;
	__syncthreads();
	int deltaSize2 = deltaSize * deltaSize;
	int tlen = deltaSize2 * batch;
	for(int i = 0; i < tlen; i += blockDim.x)
	{
		int idx = i + threadIdx.x;
		if(idx < tlen)
		{
			int s  = idx / (deltaSize2);//s
			int t2 = idx % (deltaSize2);//x,y
			int id = 
				deltaArea * c
				+ deltaSize2 * s * kernelScan2
				+ deltaSize2 * k2
				+ t2;
			_sum[threadIdx.x] += delta[id];
		}
	}
	__syncthreads();
	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < (len >> 1))
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = (len + 1) >> 1;
	}
	__syncthreads();
	if(threadIdx.x == 0)
	{
		bgrad[k2][c] = _sum[0] / batch;
	}
}